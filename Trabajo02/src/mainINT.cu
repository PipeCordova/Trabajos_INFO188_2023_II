
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <omp.h>
#include <cstdlib>

using namespace std;

// #################### Estructuras ####################
// Estructura para representar un elemento no nulo
struct ElementoNoNulo {
    int fila;
    int columna;
    int valor;
};

// Estructura para representar una matriz dispersa
struct MatrizDispersa {
    int filas;
    int columnas;
    vector<ElementoNoNulo> elementosNoNulos;
};


// #################### Funciones para imprimir ####################

// Funcion para imprimir un vector en forma de columna
void imprimeVectorCol(const vector<int>& v){
    for(int i=0 ; i<v.size() ; i++) printf("%i\n", v[i]);
}

// Función para imprimir los elementos no nulos de una matriz dispersa
void imprimirElementosNoNulos(const MatrizDispersa& matriz) {
    cout << "Elementos no nulos:" << endl;
    for (const ElementoNoNulo& elemento : matriz.elementosNoNulos) {
        cout << "(" << elemento.fila << ", " << elemento.columna << ") = " << elemento.valor << endl;
    }
}

// Función para imprimir una matriz dispersa
void imprimirMatrizDispersa(const MatrizDispersa& matriz) {
    for (int i = 0; i < matriz.filas; ++i) {
        for (int j = 0; j < matriz.columnas; ++j) {
            bool encontrado = false;
            for (const ElementoNoNulo& elemento : matriz.elementosNoNulos) {
                if (elemento.fila == i && elemento.columna == j) {
                    cout << elemento.valor << "\t";
                    encontrado = true;
                    break;
                }
            }
            if (!encontrado) {
                cout << "0\t";
            }
        }
        cout << endl;
    }
}

// #################### Funciones Matriz dispersa ####################

// Función para llenar una matriz dispersa con una densidad dada
void llenarMatrizDispersa(MatrizDispersa& matriz, double densidad) {
    // Limpiar elementos existentes
    matriz.elementosNoNulos.clear();

    // Generar elementos no nulos aleatorios
    for (int i = 0; i < matriz.filas; ++i) {
        for (int j = 0; j < matriz.columnas; ++j) {
            if ((rand() % 100) < (densidad * 100)) {
                int valor = rand() % 10 + 1;  // Valor aleatorio entre 1 y 10
                matriz.elementosNoNulos.push_back({i, j, valor});
            }
        }
    }
} 

// Función para multiplicar una matriz dispersa por un vector (Paralelismo GPU)
__global__ void multiplicarMatrizPorVectorGPU(const ElementoNoNulo* elementos, int numElementos, const int* vec, int* vectorResultadoGPU) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    while (tid < numElementos) {
        const ElementoNoNulo& elemento = elementos[tid];
        atomicAdd(&vectorResultadoGPU[elemento.fila], elemento.valor * vec[elemento.columna]);
        tid += gridDim.x * blockDim.x;
    }
}
 
// Función auxiliar para el correcto funcionamiento de la función "multiplicarMatrizPorVectorGPU"
void multiplicarMatrizPorVectorCUDA(const MatrizDispersa& matriz, const vector<int>& vec, vector<int>& vectorResultadoGPU, double& tiempo_total_GPU) {
    if (matriz.columnas != vec.size() || matriz.filas != vectorResultadoGPU.size()) {
        cerr << "Error: Las dimensiones de la matriz y el vector no son compatibles para la multiplicación." << endl;
        exit(EXIT_FAILURE);
    }

    // Copiar datos a la GPU
    ElementoNoNulo* d_elementos;
    int* d_vec;
    int* d_vectorResultadoGPU;

    hipMalloc((void**)&d_elementos, matriz.elementosNoNulos.size() * sizeof(ElementoNoNulo));
    hipMalloc((void**)&d_vec, vec.size() * sizeof(int));
    hipMalloc((void**)&d_vectorResultadoGPU, vectorResultadoGPU.size() * sizeof(int));

    hipMemcpy(d_elementos, matriz.elementosNoNulos.data(), matriz.elementosNoNulos.size() * sizeof(ElementoNoNulo), hipMemcpyHostToDevice);
    hipMemcpy(d_vec, vec.data(), vec.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vectorResultadoGPU, vectorResultadoGPU.data(), vectorResultadoGPU.size() * sizeof(int), hipMemcpyHostToDevice);

    // Configuración de bloques y hilos
    int blockSize = 256;
    int numBlocks = (matriz.elementosNoNulos.size() + blockSize - 1) / blockSize;

    // Llamada al kernel de CUDA
    double t0 = omp_get_wtime();
    multiplicarMatrizPorVectorGPU<<<numBlocks, blockSize>>>(d_elementos, matriz.elementosNoNulos.size(), d_vec, d_vectorResultadoGPU);
    double t1 = omp_get_wtime();
    tiempo_total_GPU = t1-t0;
    // Copiar resultados de vuelta a la CPU
    hipMemcpy(vectorResultadoGPU.data(), d_vectorResultadoGPU, vectorResultadoGPU.size() * sizeof(int), hipMemcpyDeviceToHost);

    // Liberar memoria en la GPU
    hipFree(d_elementos);
    hipFree(d_vec);
    hipFree(d_vectorResultadoGPU);
}

// Función para multiplicar una matriz dispersa por un vector (Parelelismo CPU)
void multiplicarMatrizPorVectorCPU(const MatrizDispersa& matriz, const vector<int>& vec, vector<int>& vectorResultadoCPU) {
    if (matriz.columnas != vec.size() || matriz.filas != vectorResultadoCPU.size()) {
        cerr << "Error: Las dimensiones de la matriz y el vector no son compatibles para la multiplicación." << endl;
        exit(EXIT_FAILURE);
    }

    #pragma omp parallel for
    for (int i = 0; i < matriz.elementosNoNulos.size(); ++i) {
        const ElementoNoNulo& elemento = matriz.elementosNoNulos[i];
        #pragma omp atomic
        vectorResultadoCPU[elemento.fila] += elemento.valor * vec[elemento.columna];
    }
}

// Función que genera un vector con números aleatorios entre 1 y 10
void generarVector(vector<int>& vec) {
    for (int i = 0; i < vec.size(); ++i) {
        vec[i] = rand() % 10 + 1; // Números aleatorios entre 1 y 10
    }
}

int main(int argc, char *argv[]) {
    if(argc != 6){
        cerr << "Ejecutar como: ./prog <n> <d> <s> <m> <nt>\nn = tamaño matriz\nd = densidad números no nulos (0 < d < 1)\ns = semilla de números aleatorios\nm = modo de ejecución\n\t-> 0 = modo CPU\n\t-> 1 = modo GPU\n\t-> 2 = modo CPU y GPU a la vez\nnt = número threads de la CPU" << endl;
        exit(EXIT_FAILURE);
    }

    system("clear");
    int n = atoi(argv[1]);
    double densidad = atof(argv[2]);
    int seed = atoi(argv[3]);
    int modo = atoi(argv[4]);
    int nt = atoi(argv[5]);
    printf("n = %i, d = %f, s = %i, m = %i, nt = %i\n", n, densidad, seed, modo, nt); fflush(stdout);
    omp_set_num_threads(nt);

    // Inicializar la semilla de números aleatorios
    srand(seed); 

    // Crear matriz dispersa
    MatrizDispersa matriz;
    matriz.filas = n;
    matriz.columnas = n;

    // Llenar matriz dispersa con números random entre 1 y 10
    printf("Llenando Matriz Dispersa......."); fflush(stdout);
    llenarMatrizDispersa(matriz, densidad);
    printf("Done!\n");fflush(stdout);

    if(n < 16) imprimirMatrizDispersa(matriz);

    // Generar vector de 1xN para multiplicar la matriz
    vector<int> vec(n);
    printf("Generando vector para multiplicar......."); fflush(stdout);
    generarVector(vec);
    printf("Done!\n");fflush(stdout);
    if(n < 16){
        printf("\nVector para multiplicar:\n"); fflush(stdout);
        imprimeVectorCol(vec);
    }

    if(modo == 0){ // MODO CPU
        // Multiplicar la matriz por el vector en modo CPU
        printf("\n\t######### MODO CPU #########");fflush(stdout);
        vector<int> vectorResultadoCPU(n,0);
        double start_time_CPU = omp_get_wtime();
        multiplicarMatrizPorVectorCPU(matriz, vec, vectorResultadoCPU);
        double end_time_CPU = omp_get_wtime();
        double tiempo_total_CPU = end_time_CPU - start_time_CPU;

        if(n < 16){
            printf("\nResultado de la multiplicación CPU:\n"); fflush(stdout);
            imprimeVectorCol(vectorResultadoCPU);
        }

        printf("\nTiempo Total modo CPU (%i threads)= %f segudos\n", nt, tiempo_total_CPU);fflush(stdout);

    } else if(modo == 1){ // MODO GPU
        printf("\n\t######### MODO GPU #########");fflush(stdout);
        // Multiplicar la matriz por el vector en modo GPU
        vector<int> vectorResultadoGPU(n,0);
        double tiempo_total_GPU;
        multiplicarMatrizPorVectorCUDA(matriz, vec, vectorResultadoGPU, tiempo_total_GPU);

        if(n < 16){
            printf("\nResultado de la multiplicación GPU:\n"); fflush(stdout);
            imprimeVectorCol(vectorResultadoGPU);
        }

        printf("\nTiempo Total modo GPU = %f segundos\n", tiempo_total_GPU);fflush(stdout);


    } else if(modo == 2){
        printf("\n\t####### MODO CPU y GPU #######");fflush(stdout);
        // Modo CPU y GPU a la vez para comprar
        // Multiplicar la matriz por el vector en modo CPU
        vector<int> vectorResultadoCPU(n,0);
        double start_time_CPU = omp_get_wtime();
        multiplicarMatrizPorVectorCPU(matriz, vec, vectorResultadoCPU);
        double end_time_CPU = omp_get_wtime();
        double tiempo_total_CPU = end_time_CPU - start_time_CPU;

        // Multiplicar la matriz por el vector en modo GPU
        vector<int> vectorResultadoGPU(n,0);
        double tiempo_total_GPU;
        multiplicarMatrizPorVectorCUDA(matriz, vec, vectorResultadoGPU, tiempo_total_GPU);

        // Imprimir el resultado
        if(n < 16){
            printf("\nResultado de la multiplicación CPU:\n"); fflush(stdout);
            imprimeVectorCol(vectorResultadoCPU);
            printf("\nResultado de la multiplicación GPU:\n"); fflush(stdout);
            imprimeVectorCol(vectorResultadoGPU);
        }

        // Imprimir tiempos totales
        printf("\nTiempo Total modo CPU (%i threads)= %f segudos\nTiempo Total modo GPU = %f segundos", nt, tiempo_total_CPU, tiempo_total_GPU);fflush(stdout);
        
        // Verificar que los resultados sean iguales
        if(vectorResultadoCPU == vectorResultadoGPU ){ printf("\n\nModo CPU y modo GPU entregan el mismo resultado!\n"); fflush(stdout);}
        else {printf("\nAlgo salió mal, dan distintos resultados\n"); fflush(stdout);}
    }

    // Imprimir los elementos no nulos y su proporción (Para verificar la densidad)
    //imprimirElementosNoNulos(matriz);
    int elemNoNulos = matriz.elementosNoNulos.size();
    float proporcion = (float)matriz.elementosNoNulos.size() / ((float)n*(float)n);
    printf("\nHay %i de %i elementos no nulos (%f densidad real)\n", elemNoNulos, n*n, proporcion); fflush(stdout);

    return 0;
}
